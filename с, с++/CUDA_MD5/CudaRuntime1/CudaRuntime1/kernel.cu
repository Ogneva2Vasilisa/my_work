#include "hip/hip_runtime.h"
﻿//#define _CRT_UNUSED
#include "hip/hip_runtime.h"
#include ""
#include <locale.h>
#include <time.h>
#include <string.h>
#include <malloc.h>
#include "md5.c"
#include <stdio.h>

FILE* slov;
char* stroka;

const unsigned long long n = 100000000;//214194124
__device__ const int len_slov = 20; //-----------------------------------!!!!!!!!!!!!!!!!!!!!!!!!!!

clock_t start_time;
clock_t end_time;
clock_t start_time1;
clock_t end_time1;

__device__ __host__ void print_hash(uint8_t* p) {
    for (unsigned int i = 0; i < 16; ++i) {
        printf("%02x", p[i]);
    }
    printf("\n");
}
__device__ __host__ void fprint_hash(FILE* file, uint8_t* p) {
    for (unsigned int i = 0; i < 16; ++i) {
        fprintf(file,"%02x", p[i]);
    }
    fprintf(file,"\n");
}

hipError_t addWithCuda( char* a, uint8_t* result, unsigned long long size);

__global__ void addKernel(char *a, uint8_t* res)
{
    long long i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n)
    {        
        //printf("%d : %d : %d\n", blockIdx.x, threadIdx.x, i);
        md5String (a + i * len_slov, res + i *16);  
        __syncthreads();
    }
}

int main()
{
    const long long arraySize = n;
    printf("kol-vo slov %lld\n", n);
    //char a[arraySize] ;
    setlocale(LC_ALL, "Ru");
    uint8_t* result;
    result = (uint8_t*)malloc(16 * n);
    stroka = (char*)malloc(len_slov*arraySize);
    memset(result, 0, 16 * n);
    long long i = 0;
    //чтение из файла и запись в строку

    {
        FILE* mf;       // Переменная, в которую поочередно будут помещаться считываемые строки
        char str[25];   //Указатель, в который будет помещен адрес массива, в который считана строка, или NULL если достигнут коней файла или произошла ошибка
        char* estr;     // Открытие файла с режимом доступа «только чтение» и привязка к нему потока данных
        printf("Открытие файла : ");
        mf = fopen("itog.txt", "r");

        // Проверка открытия файла
        if (mf == NULL) { printf("ошибка\n"); return -1; }
        else printf("выполнено\n");
        printf("Считаны строки : \n");

        //Чтение (построчно) данных из файла в бесконечном цикле
        while (i != n)
        {
            // Чтение одной строки  из файла
            estr = fgets(str, sizeof(str), mf);
            //Проверка на конец файла или ошибку чтения
            if (estr == NULL)
            {
                // Проверяем, что именно произошло: кончился файл или это ошибка чтения
                if (feof(mf) != 0)
                {
                    //Если файл закончился, выводим сообщение о завершении чтения и выходим из бесконечного цикла
                    printf("\nЧтение файла закончено\n");
                    break;
                }
                else
                {
                    //Если при чтении произошла ошибка, выводим сообщение об ошибке и выходим из бесконечного цикла
                    printf("\nОшибка чтения из файла\n");
                    break;
                }
            }
            //Если файл не закончился, и не было ошибки чтения выводим считанную строку  на экран           
            // i - итератор по кол-ву слов
            int k = 0;//итератор по готовому слову
            for (int j = 0; j < strlen(str) - 1; j++, k++)
            {
                if (str[j] != '\n') {
                    *(stroka + i * len_slov + k) = str[j];
                }
            }
            while (k < len_slov)
            {
                *(stroka + i * len_slov + k) = 0;
                k++;
            }
            i++;
        }
        // Закрываем файл
        printf("Закрытие файла : ");
        if (fclose(mf) == EOF) printf("ошибка\n");
        else printf("выполнено\n");
    }
    
    //==============================================================================================================

    printf("\n");
    start_time = clock();
    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(stroka, result, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }
    end_time = clock();
    printf("\nTime exec itog: %0.f ms\n", (double)(end_time - start_time));
    for (int i = 0; i < len_slov*arraySize; i++)
    {
        //printf("%d %c, ", stroka[i], stroka[i]);
    }

    printf("\n");
    //консоль 10 первых слов
    {
        for (i = 0; i < 10; i++)
        {
            printf("%s ", (stroka + i * len_slov));
            printf(" ");
            print_hash(result + i * 16);
        }
    }
    //запись все в файл
    printf("Запись выходных данных\n");
    FILE* out_file = fopen("output.txt", "w");
    {
        for (int i = 0; i < 10; i++) {//-------------------------------------!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
            //printf("%s ", (c + i * 20));
            fprintf(out_file, "%s ", (stroka + i * len_slov));
            fprint_hash(out_file, (result + i * 16));
        }
    }
    fclose(out_file);
    printf("Запись выполнена\n");
    
    printf("\nTime exec itog: %0.f ms\n", (double)(end_time - start_time));

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    free(result);
    free(stroka);

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(char *a, uint8_t* result, unsigned long long size)
{
    char *dev_a = 0;
    uint8_t *dev_res = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(char) * len_slov);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_res, size * sizeof(uint8_t)*16);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(char) * len_slov, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
     
    start_time1 = clock();

    const unsigned long long BLOCK_SIZE = 1024;
    //const unsigned long long BLOCK_COUNT = 10;
    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<(n + BLOCK_SIZE) / BLOCK_SIZE, BLOCK_SIZE >>>(dev_a, dev_res);//первое число
    
    end_time1 = clock();
    printf("\nTime exec: %0.f ms\n", (double)(end_time1 - start_time1));
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    cudaStatus = hipMemcpy(result, dev_res, size * sizeof(uint8_t)*16, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    

Error:
    hipFree(dev_a);
    hipFree(dev_res);
    
    return cudaStatus;
}
